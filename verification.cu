#include <cassert>
#include <fstream>
#include <iostream>
#include <string>

#include "verification.hu"

/**
 * @brief Customized assert for this use case
 *
 * @param word1 word from one file
 * @param word2 word from other file
 * @param line line number being compared
 */
void custom_assert (std::string word1, std::string word2, unsigned int line)
{
  if (word1 != word2) {
    std::string message = "ERROR: Input File and Decoded File do not match on line " + std::to_string(line);
    std::cerr << message << std::endl;
  }
}

/**
 * @brief Verifies that the decoded file is the same as the input file
 *
 * @param d_file decoded filename
 * @param i_file input filename
 * @param aNum number of words in the files
 */
void verify(const char *d_file, const char *i_file, unsigned int aNum)
{
  std::string dec_word, ref_word;

  // open the decoded file
  std::ifstream decoded_file(d_file);
  if (!decoded_file.is_open())
  {
    std::cerr << "ERROR: Unable to open decoded file for verification!" << std::endl;
  }

  // open the original input file
  std::ifstream input_file(i_file);
  if (!input_file.is_open())
  {
    std::cerr << "ERROR: Unable to open input file for verification!" << std::endl;
  }

  // verify that the decoded file is the same as the input file
  for (unsigned int i = 0; i < aNum; ++i)
  {
    std::getline(decoded_file, dec_word);
    std::getline(input_file, ref_word);

    custom_assert(dec_word, ref_word, i+1);
  }
}